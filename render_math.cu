#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "render_math.cuh"
#include "sim_math.cuh"

#define RGB_TO_INT(r, g, b) ((r << 16) | (g << 8) | b)

#ifdef __cplusplus
extern "C" {
#endif

__host__ void allocate_render_memory(int** gpu_bitmap, int_v2 size)
{
    hipMalloc(gpu_bitmap, size.x * size.y * sizeof(int));
}

__host__ void realloc_render_memory(int** gpu_bitmap, int_v2 size)
{
    hipFree(*gpu_bitmap);
    hipMalloc(gpu_bitmap, size.x * size.y * sizeof(int));
}

__host__ void free_render_memory(int* gpu_bitmap)
{
    hipFree(gpu_bitmap);
}

__host__ int* get_colored_bitmap(float_v2 *positions, int_v2 size, settings s, entry *entries, int *start_indices, int *gpu_bitmap)
{
    hipMemset(gpu_bitmap, RGB_TO_INT(255,255,255), size.x * size.y * sizeof(int));

    dim3 block_size(16,16);
    dim3 num_blocks(size.x / block_size.x, size.y / block_size.y);

    set_bitmap_colors<<<num_blocks, block_size>>>(gpu_bitmap, positions, size, s, entries, start_indices);
    hipDeviceSynchronize();

    int *bitmap = (int*)malloc(size.x * size.y * sizeof(int));
    hipMemcpy(bitmap, gpu_bitmap, size.x * size.y * sizeof(int), hipMemcpyDeviceToHost);

    return bitmap;
}

#ifdef __cplusplus
}
#endif

__global__ void set_bitmap_colors(int *gpu_bitmap, float_v2 *positions, int_v2 size, settings s, entry *entries, int *start_indices)
{
    int x = (int) (blockIdx.x * blockDim.x + threadIdx.x);
    int y = (int) (blockIdx.y * blockDim.y + threadIdx.y);

    if(x > size.x || y > size.y)
        return;

    const float_v2 pos = {(float) x * PIXELTOUNIT, (float) y * PIXELTOUNIT};

    //Absorption factors
    const float a_red = s.rs.red_absorption;
    const float a_green = s.rs.green_absorption;
    const float a_blue = s.rs.blue_absorption;

    const float h_2 = s.ss.smoothing_length * s.ss.smoothing_length;

    float_v2 calculated_density = calculate_density(positions, pos, s, entries, start_indices, {(int) ceilf( (float) size.x * 0.1f), (int) ceilf( (float) size.y * 0.1f) });

    /* Might want to do this with the normalized density. Didn't bring up any issues as far as I have seen,
     * so I left it like this
     */

    if(calculated_density.x < poly6_smoothing_kernel(h_2 * 0.1f, h_2))
    {
        gpu_bitmap[(int)(y * size.x + x)] = RGB_TO_INT(255,255,255);
        return;
    }

    /* Theoretical maximum density calculation (Really theoretical, extremely theoretical, but gives good results)
     * With this approximation, particles would have an ideal center of maximum density (distance 0), and around another "circle" of 6 particles,
     * with distance h/2 (^2 = h^2/4). Lastly we would have a final "circle" of 12 particles with distance h/sqrt(2) (^2 = h^2/2)
     * (Imagining the particles evenly distributed in the 2d space, 60 degrees apart from each other)
     */
    const float max_theoretical_density = poly6_smoothing_kernel(0.0f, h_2) +
                                          6 * poly6_smoothing_kernel(h_2 * 0.25f, h_2) +
                                          12 * poly6_smoothing_kernel(h_2 * 0.5f, h_2);

    /* Normalize the density relative to maximum theoretical density (Again, really theoretical maximum density)
     * this sadly, makes it way less "fluid-like". But I guess it's a good tradeoff for it working with most smoothing radii (or at least the ones I tested)
     */

    const float normalized_density = calculated_density.x / max_theoretical_density;

    //Damping normalized_density to spread the colors more
    const float final_light = normalized_density * 0.2f;

    //The values are clamped between 15*absorption factor and 255
    const int red = max((int)(a_red * 15), min(255, (int)(255 * final_light * a_red)));
    const int green = max((int)(a_green * 15), min(255, (int)(255 * final_light * a_green)));
    const int blue = max((int)(a_blue * 15), min(255, (int)(255 * final_light * a_blue)));

    //Color the pixel in the bitmap
    gpu_bitmap[(int)(y * size.x + x)] = RGB_TO_INT((255-red),(255-green),(255-blue));
}